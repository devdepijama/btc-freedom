﻿#include "Base58.cuh"

#include <string>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <openssl/bn.h>

using namespace std;

string Base58::cypher(char* hexBytes) {
    char table[] = { '1','2','3','4','5','6','7','8','9','A','B','C','D','E','F','G','H','J','K','L','M','N','P','Q','R','S','T','U','V','W','X','Y','Z','a','b','c','d','e','f','g','h','i','j','k','m','n','o','p','q','r','s','t','u','v','w','x','y','z' };

    //printf("Calculating b58 of %s \n", hexBytes);

    BIGNUM* base58 = NULL;

    BIGNUM* resultExp = BN_new();
    BIGNUM* resultAdd = BN_new();
    BIGNUM* resultRem = BN_new();
    BN_CTX* bn_ctx = BN_CTX_new();

    BN_dec2bn(&base58, "58");

    string endresult;
    vector<int> v;

    BN_hex2bn(&resultAdd, hexBytes);

    while (!BN_is_zero(resultAdd)) {
        BN_div(resultAdd, resultRem, resultAdd, base58, bn_ctx);
        //printf("resultAdd = %s | resultRem = %s \n", BN_bn2dec(resultRem), BN_bn2dec(resultAdd));
        v.push_back(atoi(BN_bn2dec(resultRem)));
    }

    for (int i = 0; i < strlen(hexBytes);) {
        if ((hexBytes[i] == '0') && (hexBytes[i + 1] == '0')) {
            endresult = endresult + '1'; 
            i += 2;
        }
        break;        
    }

    for (int i = (int)v.size() - 1; i >= 0; i--) {
        endresult = endresult + table[v[i]];
    }

    BN_free(resultAdd);
    BN_free(resultExp);
    BN_free(resultRem);
    BN_CTX_free(bn_ctx);

    return endresult;
}
